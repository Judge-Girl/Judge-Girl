#include "hip/hip_runtime.h"
#include "labeling.h"
#include <stdint.h>
#include <stdio.h>
#define BLK 2
#define MAXK 500
__global__ void stage_prefix(int n, const char *str, int32_t *pos) {
    extern __shared__ int32_t tmp[];
    const int m = blockDim.x;
    const int tid = threadIdx.x;
    const int base = (blockDim.x*blockIdx.x)*2;
#define TRANS(index) ((index >= n || str[index] > ' ') ? base*BLK-1 : index)
     int pout = 0, pin = 1;
    int reg_pos[BLK*2], prev;
    {
        int mx = -1, off = (2*tid+0+base)*BLK;
        for (int i = 0; i < 2*BLK; i++) {
            int t = TRANS(off+i);
            if (t > mx)    mx = t;
            reg_pos[i] = mx;
        }
        tmp[pout*m + tid] = mx;
        prev = mx;
    }
    __syncthreads();
    for (int offset = 1; offset < 256; offset <<= 1) {
        pout = 1 - pout, pin = 1 - pin;
        int t = tid >= offset ? max(tmp[pin*m + (tid-offset)], prev) : tmp[pin*m + tid];
        tmp[pout*m + tid] = prev = t;
        __syncthreads();
    }    
    {
        int off = (2*tid+0+base)*BLK;
        int mx = tid ? tmp[pout*m + tid-1] : base*BLK-1;
        for (int i = min(2*BLK, n-off)-1; i >= 0; i--) {
            pos[off+i] = off+i-max(reg_pos[i], mx);
        }
    }
}
__global__ void stage_third(int n, int32_t *pos, int bsz) {
    const int tid = threadIdx.x;
    const int base = (blockDim.x*blockIdx.x)*2;
    int l = base*BLK;
    if (l >= n || l == 0 || 2*tid*BLK >= MAXK)
        return ;
    __shared__ int fixed;
    if (tid == 0)
        fixed = pos[l-1];
    __syncthreads();
    if (fixed <= 0)
        return ;
    if (2*tid*BLK+fixed >= MAXK)
        return ;
    int off = 2*tid*BLK + l;
    int r = min(2*BLK, n-off);
    int mx = fixed+1 + 2*tid*BLK;
    for (int i = 0, j = 2*tid*BLK+1; i < r && pos[off] == j; i++, j++) {
        pos[off] = mx+i, off++;
    }
}
void labeling(const char *cuStr, int32_t *cuPos, int strLen) {
    const int bsz = 256;
    int M = bsz*BLK*2;
    dim3 bb(bsz);
    dim3 gg((strLen+M-1)/M);
    stage_prefix<<<gg, bb, (bsz*2+1)*sizeof(int32_t)>>>(strLen, cuStr, cuPos);
    stage_third<<<gg, bb>>>(strLen, cuPos, bsz*2*BLK);
}
